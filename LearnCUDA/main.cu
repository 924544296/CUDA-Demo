#include "LearnCUDA.cuh"


//#define T int
using T = long long;


int main()
{
	/* ��ӡCUDA�汾 */
	int runtimeVersion = 0;
	CHECK(hipRuntimeGetVersion(&runtimeVersion)); 
	int driverVersion = 0;
	CHECK(hipDriverGetVersion(&driverVersion));
	std::cout << "CUDA Runtime Version: " << runtimeVersion / 1000 << "." << (runtimeVersion % 1000) / 10 << std::endl; 
	std::cout << "CUDA Driver Version: " << driverVersion / 1000 << "." << (driverVersion % 1000) / 10 << std::endl;
	std::cout << "CUDA Compiler Version: " << __CUDACC_VER_MAJOR__ << "." << __CUDACC_VER_MINOR__ << std::endl;



	/* �˺����д�ӡ��Ϣ */
	const dim3 grid_size(2, 3, 4);
	const dim3 block_size(1, 1, 2);
	//hello_from_gpu <<<1, block_size>>>();
	//hello_from_gpu <<<grid_size, 1>>> ();
	hello_from_gpu <<<grid_size, block_size >>> ();
	CHECK(hipGetLastError());
	CHECK(hipDeviceSynchronize());
	hipDeviceSynchronize();



	/* ��������� */
	//const int N = 100000000;
	//const int M = sizeof(T) * N;
	//T* h_a = (T*)malloc(M);
	//T* h_b = (T*)malloc(M);
	//T* h_c = (T*)malloc(M);

	//for (int n = 0; n < N; n++)
	//{
	//	h_a[n] = (T)n;
	//	h_b[n] = 2 * (T)n;
	//}

	//T *d_a, *d_b, *d_c;
	////hipMalloc((void**)&d_a, M);
	////hipMalloc((void**)&d_b, M);
	////hipMalloc((void**)&d_c, M);
	//hipMalloc(&d_a, M);
	//hipMalloc(&d_b, M);
	//hipMalloc(&d_c, M);

	//hipMemcpy(d_a, h_a, M, hipMemcpyHostToDevice);
	//hipMemcpy(d_b, h_b, M, hipMemcpyHostToDevice);

	//const dim3 grid_size(N / 1024, 4, 2);
	//const dim3 block_size(16, 4, 2);
	////add_1d<<<grid_size, block_size>>>(d_a, d_b, d_c, N);
	////add_2d<<<grid_size, block_size>>>(d_a, d_b, d_c, N);
	////add_3d<<<grid_size, block_size>>>(d_a, d_b, d_c, N);
	////add_1d_loop<<<grid_size, block_size>>>(d_a, d_b, d_c, N);
	////add_2d_loop<<<grid_size, block_size>>>(d_a, d_b, d_c, N);
	//add_3d_loop<<<grid_size, block_size>>>(d_a, d_b, d_c, N);
	//CHECK(hipGetLastError());
	//CHECK(hipDeviceSynchronize());

	//hipMemcpy(h_c, d_c, M, hipMemcpyDeviceToHost);
	//std::cout << "h_c[0] = " << h_c[0] << std::endl;
	//std::cout << "h_c[N-1] = " << h_c[N-1] << std::endl;

	//free(h_a);
	//free(h_b);
	//free(h_c);
	//hipFree(d_a);
	//hipFree(d_b);
	//hipFree(d_c);



	/* ��� */
	//const int N = 10000;
	//dim3 grid_size(64, 1, 1);
	//dim3 block_size(128, 1, 1);
	//T *h_a, *h_b, *h_c, *d_a, *d_b, *d_c, h_c_sum;

	//h_a = new T[N];
	//h_b = new T[N];
	//h_c = new T[grid_size.x * grid_size.y * grid_size.z];

	//CHECK(hipMalloc((void**)&d_a, N * sizeof(T)));
	//CHECK(hipMalloc((void**)&d_b, N * sizeof(T)));
	//CHECK(hipMalloc((void**)&d_c, grid_size.x * grid_size.y * grid_size.z * sizeof(T)));

	//for (int i = 0; i < N; i++)
	//{
	//	h_a[i] = (T)i;
	//	h_b[i] = (T)i*2;
	//}

	//CHECK(hipMemcpy(d_a, h_a, N * sizeof(T), hipMemcpyHostToDevice));
	//CHECK(hipMemcpy(d_b, h_b, N * sizeof(T), hipMemcpyHostToDevice));

	//dot_product<<<grid_size, block_size, block_size.x * block_size.y * block_size.z * sizeof(T)>>>(d_a, d_b, d_c, N);
	//CHECK(hipGetLastError());
	//CHECK(hipDeviceSynchronize());
	//hipDeviceSynchronize();

	//CHECK(hipMemcpy(h_c, d_c, grid_size.x * grid_size.y * grid_size.z * sizeof(T), hipMemcpyDeviceToHost));

	//std::cout << "h_c[0] = " << h_c[0] << std::endl;
	//std::cout << "h_c[N-1] = " << h_c[grid_size.x * grid_size.y * grid_size.z - 1] << std::endl;
	//h_c_sum = 0;
	//for (int i = 0; i < grid_size.x * grid_size.y * grid_size.z; i++)
	//{
	//	//std::cout << "h_c[" << i << "] = " << h_c[i] << std::endl;
	//	h_c_sum += h_c[i];
	//}
	//std::cout << "h_c_sum = " << h_c_sum << std::endl;

	//CHECK(hipFree(d_a));
	//CHECK(hipFree(d_b));
	//CHECK(hipFree(d_c));
	//delete[] h_a;
	//delete[] h_b;
	//delete[] h_c;


	/* ��Լ */


	std::cout << std::endl << "---->Done." << std::endl;
	return 0;
}