#include "hip/hip_runtime.h"
﻿#include "LearnCUDA.cuh"


__global__ void hello_from_gpu()
{
	printf("Hello World from block - (%d %d %d), thread - (%d %d %d)!\n", blockIdx.x, blockIdx.y, blockIdx.z, threadIdx.x, threadIdx.y, threadIdx.z);
}

